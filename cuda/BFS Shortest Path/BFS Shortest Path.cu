#include <hip/hip_runtime.h>
#include <algorithm>

__global__ void bfs_kernel(const int* input_grid, int* distances, int rows, int cols, 
                          const int* current_frontier, int current_size, 
                          int* next_frontier, int* next_size, int current_level) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= current_size) return;

    int node = current_frontier[idx];
    int row = node / cols;
    int col = node % cols;

    int dr[] = {1, -1, 0, 0};
    int dc[] = {0, 0, 1, -1};

    for (int d = 0; d < 4; d++) {
        int r = row + dr[d];
        int c = col + dc[d];
        if (r >= 0 && r < rows && c >= 0 && c < cols) {
            int index = r * cols + c;
            if (input_grid[index] == 0) {
                int expected = -1;
                if (atomicCAS(&distances[index], expected, current_level + 1) == -1) {
                    int pos = atomicAdd(next_size, 1);
                    next_frontier[pos] = index;
                }
            }
        }
    }
}

extern "C" void solve(const int* grid, int* result, int rows, int cols, 
                     int start_row, int start_col, int end_row, int end_col) {
    if (start_row == end_row && start_col == end_col) {
        int zero = 0;
        hipMemcpy(result, &zero, sizeof(int), hipMemcpyHostToDevice);
        return;
    }

    int* distances = nullptr;
    int* frontier1 = nullptr;
    int* frontier2 = nullptr;
    int* d_next_size = nullptr;

    hipMalloc(&distances, rows * cols * sizeof(int));
    hipMemset(distances, -1, rows * cols * sizeof(int));

    hipMalloc(&frontier1, rows * cols * sizeof(int));
    hipMalloc(&frontier2, rows * cols * sizeof(int));
    hipMalloc(&d_next_size, sizeof(int));

    int start_index = start_row * cols + start_col;
    hipMemset(&distances[start_index], 0, sizeof(int));

    hipMemcpy(frontier1, &start_index, sizeof(int), hipMemcpyHostToDevice);

    int current_size = 1;
    int* current_frontier = frontier1;
    int* next_frontier = frontier2;
    int level = 0;

    int end_index = end_row * cols + end_col;
    int found = 0;
    int answer = -1;

    while (current_size > 0) {
        hipMemset(d_next_size, 0, sizeof(int));

        dim3 block(256);
        dim3 grid_dim((current_size + 255) / 256);
        bfs_kernel<<<grid_dim, block>>>(grid, distances, rows, cols, current_frontier, current_size, next_frontier, d_next_size, level);
        hipDeviceSynchronize();

        int end_distance;
        hipMemcpy(&end_distance, &distances[end_index], sizeof(int), hipMemcpyDeviceToHost);
        if (end_distance != -1) {
            answer = end_distance;
            found = 1;
            break;
        }

        int next_size_val;
        hipMemcpy(&next_size_val, d_next_size, sizeof(int), hipMemcpyDeviceToHost);
        std::swap(current_frontier, next_frontier);
        current_size = next_size_val;
        level++;
    }

    if (found) {
        hipMemcpy(result, &answer, sizeof(int), hipMemcpyHostToDevice);
    } else {
        int minus_one = -1;
        hipMemcpy(result, &minus_one, sizeof(int), hipMemcpyHostToDevice);
    }

    hipFree(distances);
    hipFree(frontier1);
    hipFree(frontier2);
    hipFree(d_next_size);
}