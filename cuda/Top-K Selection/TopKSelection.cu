// https://leetgpu.com/challenges/top-k-selection

#include "solve.h"
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

// input, output are device pointers
void solve(const float* input, float* output, int N, int k) {
    float *d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;
    float *d_sorted;
    hipMalloc(&d_sorted, N * sizeof(float));

    hipcub::DeviceRadixSort::SortKeysDescending(d_temp_storage, temp_storage_bytes, input, d_sorted, N);
    hipMalloc(&d_temp_storage, temp_storage_bytes);

    hipcub::DeviceRadixSort::SortKeysDescending(d_temp_storage, temp_storage_bytes, input, d_sorted, N);
    hipMemcpy(output, d_sorted, k * sizeof(float), hipMemcpyDeviceToDevice);
    
    hipFree(d_sorted);
    hipFree(d_temp_storage);
    hipDeviceSynchronize();
}