#include "hip/hip_runtime.h"
https://leetgpu.com/challenges/3d-convolution

```cpp
#include <cstdio>
#include <hip/hip_runtime.h>

__global__ void conv3d_kernel(const float* input, const float* kernel, float* output,
                              int input_depth, int input_rows, int input_cols,
                              int kernel_depth, int kernel_rows, int kernel_cols,
                              int output_depth, int output_rows, int output_cols) {
    // 获取当前线程对应的输出坐标(i, j, k)
    int i = blockIdx.z * blockDim.z + threadIdx.z;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= output_depth || j >= output_rows || k >= output_cols) {
        return;
    }

    float sum = 0.0f;

    // 遍历kernel的三个维度
    for (int d = 0; d < kernel_depth; ++d) {
        for (int r = 0; r < kernel_rows; ++r) {
            for (int c = 0; c < kernel_cols; ++c) {
                // 计算输入元素的坐标
                int input_d = i + d;
                int input_r = j + r;
                int input_c = k + c;

                // 计算输入和kernel的线性索引
                int input_idx = input_d * (input_rows * input_cols) + input_r * input_cols + input_c;
                int kernel_idx = d * (kernel_rows * kernel_cols) + r * kernel_cols + c;

                sum += input[input_idx] * kernel[kernel_idx];
            }
        }
    }

    // 计算输出的线性索引并存储结果
    int output_idx = i * (output_rows * output_cols) + j * output_cols + k;
    output[output_idx] = sum;
}

void solve(const float* input, const float* kernel, float* output,
           int input_depth, int input_rows, int input_cols,
           int kernel_depth, int kernel_rows, int kernel_cols) {
    // 计算输出尺寸
    int output_depth = input_depth - kernel_depth + 1;
    int output_rows = input_rows - kernel_rows + 1;
    int output_cols = input_cols - kernel_cols + 1;

    if (output_depth <= 0 || output_rows <= 0 || output_cols <= 0) {
        return; // 无效输出尺寸
    }

    // 分配设备内存
    float *d_input, *d_kernel, *d_output;
    size_t input_size = input_depth * input_rows * input_cols;
    size_t kernel_size = kernel_depth * kernel_rows * kernel_cols;
    size_t output_size = output_depth * output_rows * output_cols;

    hipMalloc(&d_input, input_size * sizeof(float));
    hipMalloc(&d_kernel, kernel_size * sizeof(float));
    hipMalloc(&d_output, output_size * sizeof(float));

    // 拷贝数据到设备
    hipMemcpy(d_input, input, input_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, kernel, kernel_size * sizeof(float), hipMemcpyHostToDevice);

    // 配置线程块和网格
    dim3 block(8, 8, 4); // 线程块维度
    dim3 grid(
        (output_cols + block.x - 1) / block.x,
        (output_rows + block.y - 1) / block.y,
        (output_depth + block.z - 1) / block.z
    );

    // 启动核函数
    conv3d_kernel<<<grid, block>>>(d_input, d_kernel, d_output,
        input_depth, input_rows, input_cols,
        kernel_depth, kernel_rows, kernel_cols,
        output_depth, output_rows, output_cols);

    // 检查CUDA错误
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
    }

    // 拷贝结果回主机
    hipMemcpy(output, d_output, output_size * sizeof(float), hipMemcpyDeviceToHost);

    // 释放设备内存
    hipFree(d_input);
    hipFree(d_kernel);
    hipFree(d_output);
}
```