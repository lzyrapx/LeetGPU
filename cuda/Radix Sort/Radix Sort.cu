#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

// input, output are device pointers
extern "C" void solve(const unsigned int* input, unsigned int* output, int N) {
    void* d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;
    
    hipcub::DeviceRadixSort::SortKeys(
        d_temp_storage, 
        temp_storage_bytes, 
        input, 
        output, 
        N
    );
    
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    
    hipcub::DeviceRadixSort::SortKeys(
        d_temp_storage, 
        temp_storage_bytes, 
        input, 
        output, 
        N,
        0,      // begin_bit
        32      // end_bit
    );
    
    hipFree(d_temp_storage);       
}