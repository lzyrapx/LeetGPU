#include "hip/hip_runtime.h"
// https://leetgpu.com/challenges/matrix-power

#include "solve.h"
#include <hip/hip_runtime.h>

__global__ void identityKernel(float *mat, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < N && col < N) {
        mat[row * N + col] = (row == col) ? 1.0f : 0.0f;
    }
}

__global__ void matrixMultiplyKernel(const float *A, const float *B, float *C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < N && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < N; k++) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

void solve(const float* input, float* output, int N, int P) {
    if (P == 0) {
        dim3 threads(16, 16);
        dim3 blocks((N + 15) / 16, (N + 15) / 16);
        identityKernel<<<blocks, threads>>>(output, N);
        hipDeviceSynchronize();
        return;
    }

    if (P == 1) {
        hipMemcpy(output, input, N * N * sizeof(float), hipMemcpyDeviceToDevice);
        return;
    }

    float *d_buf1, *d_buf2, *d_buf3;
    hipMalloc(&d_buf1, N * N * sizeof(float));
    hipMalloc(&d_buf2, N * N * sizeof(float));
    hipMalloc(&d_buf3, N * N * sizeof(float));

    dim3 threads(16, 16);
    dim3 blocks((N + 15) / 16, (N + 15) / 16);

    identityKernel<<<blocks, threads>>>(d_buf1, N);
    hipMemcpy(d_buf2, input, N * N * sizeof(float), hipMemcpyDeviceToDevice);
    hipDeviceSynchronize();

    float *d_result = d_buf1;
    float *d_base = d_buf2;
    float *d_temp = d_buf3;

    int exponent = P;
    while (exponent) {
        if (exponent & 1) {
            matrixMultiplyKernel<<<blocks, threads>>>(d_result, d_base, d_temp, N);
            hipDeviceSynchronize();
            float *tmp = d_result;
            d_result = d_temp;
            d_temp = tmp;
        }
        exponent >>= 1;
        if (exponent) {
            matrixMultiplyKernel<<<blocks, threads>>>(d_base, d_base, d_temp, N);
            hipDeviceSynchronize();
            float *tmp = d_base;
            d_base = d_temp;
            d_temp = tmp;
        }
    }

    hipMemcpy(output, d_result, N * N * sizeof(float), hipMemcpyDeviceToDevice);

    hipFree(d_buf1);
    hipFree(d_buf2);
    hipFree(d_buf3);
}