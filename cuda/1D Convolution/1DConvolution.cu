#include "hip/hip_runtime.h"

// https://leetgpu.com/challenges/1d-convolution

#include "solve.h"
#include <hip/hip_runtime.h>

__global__ void convolution_1d_kernel(const float* input, const float* kernel, float* output,
                                      int input_size, int kernel_size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int output_size = input_size - kernel_size + 1;
    if (i >= output_size) return;

    float sum = 0.0f;
    for (int j = 0; j < kernel_size; ++j) {
        sum += input[i + j] * kernel[j];
    }
    output[i] = sum;
}

// input, kernel, output are device pointers (i.e. pointers to memory on the GPU)
void solve(const float* input, const float* kernel, float* output, int input_size, int kernel_size) {
    int output_size = input_size - kernel_size + 1;
    int threadsPerBlock = 256;
    int blocksPerGrid = (output_size + threadsPerBlock - 1) / threadsPerBlock;

    convolution_1d_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, kernel, output, input_size, kernel_size);
    hipDeviceSynchronize();
}