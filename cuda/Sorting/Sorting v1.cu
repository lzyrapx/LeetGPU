#include "solve.h"
#include <cub/device/device_radix_sort.cuh>

void solve(float* data, int N) {
    size_t temp_storage_bytes = 0;
    hipcub::DeviceRadixSort::SortKeys(nullptr, temp_storage_bytes, data, data, N);
    
    void* d_temp_storage = nullptr;
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    
    // 升序
    hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, data, data, N);
    
    hipFree(d_temp_storage);
}