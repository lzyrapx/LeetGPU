#include <hip/hip_runtime.h>

__global__ void count_2d_equal_kernel(const int* input, int* output, int N, int M, int K) {
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    if (col < M && row < N && input[row * M + col] == K) {
        atomicAdd(output, 1);
    }
}

// input, output are device pointers (i.e. pointers to memory on the GPU)
extern "C" void solve(const int* input, int* output, int N, int M, int K) {
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((M + threadsPerBlock.x - 1) / threadsPerBlock.x,
                              (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    count_2d_equal_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, output, N, M, K);
    hipDeviceSynchronize();
}